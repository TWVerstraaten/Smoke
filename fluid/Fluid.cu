#include "hip/hip_runtime.h"
#include "Fluid.cuh"

#include <cassert>

namespace app {
    namespace fluid {

        // See Jos Stam's paper "Real-Time Fluid Dynamics for Games" for the algorithm below

        static const unsigned g_threads = 1024;
        static const unsigned g_blocks  = (g_point_count * g_point_count + g_threads - 1) / g_threads;
        __constant__ unsigned g_n       = g_point_count * g_point_count;

        struct Index {
            uint16_t i;
            uint16_t j;
        };

        static Index screen_to_array_indices(float x, float y);
        static float clamp_to_zero_one(float val);

        float Fluid::sample_density_at(float x, float y) const {
            return m_density.sample_at(x, y);
        }

        float Fluid::sample_u_at(float x, float y) const {
            return m_u.sample_at(x, y);
        }

        float Fluid::sample_v_at(float x, float y) const {
            return m_v.sample_at(x, y);
        }

        __device__ int diff(unsigned a, unsigned b) {
            return a >= b ? static_cast<int>(a - b) : -static_cast<int>(b - a);
        }

        __global__ void add_around_point_kernel(float* density, uint16_t i_target, uint16_t j_target, float multiplier, float radius) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || i == g_point_count || j == g_point_count)
                return;

            const int d_i = diff(i, i_target);
            const int d_j = diff(j, j_target);

            const float dist = std::sqrt(static_cast<float>(d_i * d_i + d_j * d_j)) / static_cast<float>(g_point_count);
            if (dist > radius)
                return;

            density[index] += (radius - dist) * multiplier / radius;
        }

        void Fluid::add_density(float x, float y, float multiplier) {
            if (x <= 0 || x >= 1 || y <= 0 || y >= 1) {
                return;
            }
            const auto grid_position = screen_to_array_indices(x, y);
            add_around_point_kernel<<<g_blocks, g_threads>>>(m_density_gpu, grid_position.i, grid_position.j, multiplier * g_particle_input, 0.035);
            hipDeviceSynchronize();
        }

        void Fluid::add_velocity(float x, float y, float dx, float dy) {
            if (x <= 0 || x >= 1 || y <= 0 || y >= 1) {
                return;
            }
            const auto grid_position = screen_to_array_indices(x, y);
            add_around_point_kernel<<<g_blocks, g_threads>>>(m_u_previous_gpu, grid_position.i, grid_position.j, g_force_input * dx, 0.035);
            add_around_point_kernel<<<g_blocks, g_threads>>>(m_v_previous_gpu, grid_position.i, grid_position.j, g_force_input * dy, 0.035);
            hipDeviceSynchronize();
        }

        __global__ void set_to_zero_kernel(float* data) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            data[index] = 0;
        }

        void Fluid::clear_previous() {
            set_to_zero_kernel<<<g_blocks, g_threads>>>(m_density_previous_gpu);
            set_to_zero_kernel<<<g_blocks, g_threads>>>(m_v_previous_gpu);
            set_to_zero_kernel<<<g_blocks, g_threads>>>(m_u_previous_gpu);
            hipDeviceSynchronize();
        }

        void Fluid::clear_current() {
            set_to_zero_kernel<<<g_blocks, g_threads>>>(m_density_gpu);
            set_to_zero_kernel<<<g_blocks, g_threads>>>(m_v_gpu);
            set_to_zero_kernel<<<g_blocks, g_threads>>>(m_u_gpu);
            hipDeviceSynchronize();
        }

        __global__ void decrease_density_kernel(float* density) {
            unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index < g_n)
                density[index] *= 0.98;
        }

        __device__ uint32_t to_linear_index(uint16_t i, uint16_t j) {
            return i * g_point_count + j;
        }

        __global__ void advect_kernel(float* current, const float* previous, const float* u, const float* v, float ratio) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;

            const float x_temp    = static_cast<float>(i) - ratio * u[index];
            const float y_temp    = static_cast<float>(j) - ratio * v[index];
            const float upper     = static_cast<float>(g_cell_count) - 0.5f;
            const float x_new     = x_temp < 0.5f ? 0.5f : x_temp > upper ? upper : x_temp;
            const float y_new     = y_temp < 0.5f ? 0.5f : y_temp > upper ? upper : y_temp;
            const int   x_new_int = static_cast<int>(x_new);
            const int   y_new_int = static_cast<int>(y_new);
            const float dt_x      = x_new - static_cast<float>(x_new_int);
            const float dt_y      = y_new - static_cast<float>(y_new_int);
            current[index]        = (1 - dt_x) * ((1 - dt_y) * previous[to_linear_index(x_new_int, y_new_int)] + dt_y * previous[to_linear_index(x_new_int, y_new_int + 1)]) +
                             dt_x * ((1 - dt_y) * previous[to_linear_index(x_new_int + 1, y_new_int)] + dt_y * previous[to_linear_index(x_new_int + 1, y_new_int + 1)]);
        }

        __global__ void set_bounds_to_zero_kernel(float* data) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;
            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || i == g_point_count - 1 || j == g_point_count - 1)
                data[index] = 0;
        }

        __device__ float sum_neighbors_kernel(const float* data, uint16_t i, uint16_t j) {
            assert(i > 0 && j > 0);
            return data[to_linear_index(i - 1, j)] + data[to_linear_index(i + 1, j)] + data[to_linear_index(i, j - 1)] + data[to_linear_index(i, j + 1)];
        }

        __global__ void diffuse_kernel(float* current, const float* previous, float ratio) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
                return;

            current[index] = (previous[index] + ratio * sum_neighbors_kernel(current, i, j)) / (1 + (4 * ratio));
        }

        __global__ void add_scaled_kernel(float* target, const float* addend, const float multiplier) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            target[index] += multiplier * addend[index];
        }

        static void advect(float* current, const float* previous, const float* u, const float* v) {
            advect_kernel<<<g_blocks, g_threads>>>(current, previous, u, v, g_dt * static_cast<float>(g_cell_count - 1));
            hipDeviceSynchronize();

            set_bounds_to_zero_kernel<<<g_blocks, g_threads>>>(current);
            hipDeviceSynchronize();
        }

        static void diffuse(float* current, const float* previous, float weight) {
            float ratio = weight * g_diffusion_coefficient * (g_cell_count - 1.0f) * (g_cell_count - 1.0f);
            for (size_t k = 0; k != 20; ++k) {
                diffuse_kernel<<<g_blocks, g_threads>>>(current, previous, ratio);
                hipDeviceSynchronize();
            }

            set_bounds_to_zero_kernel<<<g_blocks, g_threads>>>(current);
            hipDeviceSynchronize();
        }

        static void add_scaled(float* target, const float* addend, const float multiplier) {
            add_scaled_kernel<<<g_blocks, g_threads>>>(target, addend, multiplier);
            hipDeviceSynchronize();
        }

        void Fluid::density_step() {
            add_scaled(m_density_gpu, m_density_previous_gpu, g_dt);
            diffuse(m_density_previous_gpu, m_density_gpu, g_dt);
            advect(m_density_gpu, m_density_previous_gpu, m_u_gpu, m_v_gpu);

            decrease_density_kernel<<<g_blocks, g_threads>>>(m_density_gpu);
            hipDeviceSynchronize();
        }

        __device__ float horizontal_difference(const float* data, uint16_t i, uint16_t j) {
            return data[to_linear_index(i, j + 1)] - data[to_linear_index(i, j - 1)];
        }

        __device__ float vertical_difference(const float* source, uint16_t i, uint16_t j) {
            return source[to_linear_index(i + 1, j)] - source[to_linear_index(i - 1, j)];
        }

        __global__ void project_step_3_kernel(float* u_current, float* v_current, const float* u_previous) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
                return;

            u_current[to_linear_index(i, j)] -= 0.5f * vertical_difference(u_previous, i, j) / g_cell_length;
            v_current[to_linear_index(i, j)] -= 0.5f * horizontal_difference(u_previous, i, j) / g_cell_length;
        }

        __global__ void project_step_2_kernel(float* u_previous) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
                return;

            u_previous[to_linear_index(i, j)] = sum_neighbors_kernel(u_previous, i, j) / 4.0f;
        }

        __global__ void project_step_1_kernel(float* v_previous, const float* u_current, const float* v_current) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
                return;

            v_previous[index] = -0.5f * g_cell_length * (vertical_difference(u_current, i, j) + horizontal_difference(v_current, i, j));
        }

        void project(float* u_gpu, float* v_gpu, float* u_previous_gpu, float* v_previous_gpu) {
            set_to_zero_kernel<<<g_blocks, g_threads>>>(u_previous_gpu);
            hipDeviceSynchronize();

            project_step_1_kernel<<<g_blocks, g_threads>>>(v_previous_gpu, u_gpu, v_gpu);
            hipDeviceSynchronize();

            for (size_t k = 0; k != 20; ++k) {
                project_step_2_kernel<<<g_blocks, g_threads>>>(u_previous_gpu);
                hipDeviceSynchronize();

                add_scaled(u_previous_gpu, v_previous_gpu, 0.25);
            }

            project_step_3_kernel<<<g_blocks, g_threads>>>(u_gpu, v_gpu, u_previous_gpu);
            hipDeviceSynchronize();

            set_bounds_to_zero_kernel<<<g_blocks, g_threads>>>(u_gpu);
            set_bounds_to_zero_kernel<<<g_blocks, g_threads>>>(v_gpu);
            hipDeviceSynchronize();
        }

        void Fluid::velocity_step() {
            add_scaled(m_u_gpu, m_u_previous_gpu, g_dt);
            add_scaled(m_v_gpu, m_v_previous_gpu, g_dt);
            diffuse(m_u_previous_gpu, m_u_gpu, g_viscosity_coefficient);
            diffuse(m_v_previous_gpu, m_v_gpu, g_viscosity_coefficient);
            project(m_u_previous_gpu, m_v_previous_gpu, m_u_gpu, m_v_gpu);
            advect(m_u_gpu, m_u_previous_gpu, m_u_previous_gpu, m_v_previous_gpu);
            advect(m_v_gpu, m_v_previous_gpu, m_u_previous_gpu, m_v_previous_gpu);
            project(m_u_gpu, m_v_gpu, m_u_previous_gpu, m_v_previous_gpu);
        }

        void Fluid::step() {
            velocity_step();
            density_step();
            get_from_gpu();
        }

        void Fluid::send_to_gpu() {
            hipMemcpy(m_density_gpu, m_density.data(), g_point_count * g_point_count * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(m_density_previous_gpu, m_density_previous.data(), g_point_count * g_point_count * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(m_u_gpu, m_u.data(), g_point_count * g_point_count * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(m_v_gpu, m_v.data(), g_point_count * g_point_count * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(m_u_previous_gpu, m_u_previous.data(), g_point_count * g_point_count * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(m_v_previous_gpu, m_v_previous.data(), g_point_count * g_point_count * sizeof(float), hipMemcpyHostToDevice);
        }

        void Fluid::get_from_gpu() {
            hipMemcpy(m_density.data(), m_density_gpu, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(m_density_previous.data(), m_density_previous_gpu, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(m_u.data(), m_u_gpu, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(m_v.data(), m_v_gpu, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(m_u_previous.data(), m_u_previous_gpu, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(m_v_previous.data(), m_v_previous_gpu, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
        }

        void Fluid::add_bar() {
            for (size_t i = 0; i != 20; ++i) {
                float r = static_cast<float>(rand()) / static_cast<float>(6.0f * RAND_MAX);
                //            auto [i_min, j_min] = screen_to_array_indices(static_cast<float>(i) / 20.0f, 0.0f);
                //            auto [i_max, j_max] = screen_to_array_indices(static_cast<float>(i + 1) / 2.0f, r);
                //            assert(i_min <= i_max);
                //            assert(j_min <= j_max);
                //            const float du = g_force_input * (static_cast<float>(rand()) / (92230.0f * RAND_MAX)) * ((rand() % 10) > 4 ? -1.0f : 1.0f);
                //            for (size_t w = i_min; w <= i_max; ++w) {
                //                for (size_t h = j_min; h <= j_max; ++h) {
                //                    m_density_previous[w][h] = 0.01f * g_particle_input;
                //                    m_v_previous[w][h]       = 0.0001f * g_force_input * r;
                //                    m_u_previous[w][h]       = du;
                //                }
                //            }
            }
        }

        void Fluid::set_circle() {
            for (size_t i = 0; i != g_point_count - 1; ++i) {
                for (size_t j = 0; j != g_point_count - 1; ++j) {
                    //                if (std::sqrt(tools::math::square(i - g_cell_count / 2) + tools::math::square(j - g_cell_count / 2)) < g_cell_count / 4.0f &&
                    //                    std::sqrt(tools::math::square(i - g_cell_count / 2) + tools::math::square(j - g_cell_count / 2)) > g_cell_count / 4.0f - 4) {
                    //                    m_density[i][j] = 120;
                    //                    m_u[i][j] *= 3;
                    //                    m_v[i][j] *= 3;
                    //                }
                }
            }
        }

        Fluid::Fluid() {
            hipMalloc(&m_density_gpu, g_point_count * g_point_count * sizeof(float));
            hipMalloc(&m_density_previous_gpu, g_point_count * g_point_count * sizeof(float));
            hipMalloc(&m_u_gpu, g_point_count * g_point_count * sizeof(float));
            hipMalloc(&m_v_gpu, g_point_count * g_point_count * sizeof(float));
            hipMalloc(&m_u_previous_gpu, g_point_count * g_point_count * sizeof(float));
            hipMalloc(&m_v_previous_gpu, g_point_count * g_point_count * sizeof(float));
        }

        Fluid::~Fluid() {
            hipFree(m_density_gpu);
            hipFree(m_density_previous_gpu);
            hipFree(m_u_gpu);
            hipFree(m_v_gpu);
            hipFree(m_u_previous_gpu);
            hipFree(m_v_previous_gpu);
        }
        /******************** Static function implementations ************************/

        static float clamp_to_zero_one(float val) {
            return val < 0.0f ? 0.0f : val > 1.0f ? 1.0f : val;
        }

        static Index screen_to_array_indices(float x, float y) {
            const auto i = static_cast<uint16_t>(clamp_to_zero_one(x) * (g_point_count - 1));
            const auto j = static_cast<uint16_t>(clamp_to_zero_one(y) * (g_point_count - 1));
            return {i, j};
        }
    } // namespace fluid
} // namespace app
