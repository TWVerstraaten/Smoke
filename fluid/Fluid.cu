#include "hip/hip_runtime.h"
#include "Fluid.cuh"

#include <cassert>
#include <iostream>
#include <stdint.h>

namespace app {
    namespace fluid {

        // See Jos Stam's paper "Real-Time Fluid Dynamics for Games" for the algorithm below

        static const unsigned g_threads = 128;
        static const unsigned g_blocks  = (g_point_count * g_point_count + g_threads - 1) / g_threads;
        __constant__ unsigned g_n       = g_point_count * g_point_count;

        struct Index {
            uint16_t i;
            uint16_t j;
        };

        /******************** Static function declarations ************************/
        static Index screen_to_array_indices(float x, float y);
        static float clamp_to_zero_one(float val);
        static void  advect(float* current, const float* previous, const float* u, const float* v);
        static void  diffuse(float* current, const float* previous, float weight);
        static void  add_scaled(float* target, const float* addend, float multiplier);
        static void  project(float* u_gpu, float* v_gpu, float* u_previous_gpu, float* v_previous_gpu);

        /******************** CUDA kernel function declarations ************************/
        __global__ void     add_around_point_kernel(float* density, uint16_t i_target, uint16_t j_target, float multiplier, float radius);
        __global__ void     set_to_zero_kernel(float* data);
        __global__ void     decrease_density_kernel(float* density);
        __global__ void     advect_kernel(float* current, const float* previous, const float* u, const float* v, float ratio);
        __global__ void     set_bounds_to_zero_kernel(float* data);
        __global__ void     diffuse_kernel(float* current, const float* previous, float ratio);
        __global__ void     add_scaled_kernel(float* target, const float* addend, float multiplier);
        __global__ void     project_step_3_kernel(float* u_current, float* v_current, const float* u_previous);
        __global__ void     set_circle_kernel(float* density, float* u, float* v);
        __global__ void     project_step_2_kernel(float* u_previous);
        __global__ void     project_step_1_kernel(float* v_previous, const float* u_current, const float* v_current);
        __device__ float    distance_kernel(uint16_t i_1, uint16_t j_1, uint16_t i_2, uint16_t j_2);
        __device__ float    sum_neighbors_kernel(const float* data, uint16_t i, uint16_t j);
        __device__ float    horizontal_difference(const float* data, uint16_t i, uint16_t j);
        __device__ float    vertical_difference(const float* source, uint16_t i, uint16_t j);
        __device__ uint32_t to_linear_index(uint16_t i, uint16_t j);

        /******************** Fluid member function implementations ************************/
        Fluid::Fluid() {
            hipMalloc(&m_density_gpu, g_point_count * g_point_count * sizeof(float));
            hipMalloc(&m_density_previous_gpu, g_point_count * g_point_count * sizeof(float));
            hipMalloc(&m_u_gpu, g_point_count * g_point_count * sizeof(float));
            hipMalloc(&m_v_gpu, g_point_count * g_point_count * sizeof(float));
            hipMalloc(&m_u_previous_gpu, g_point_count * g_point_count * sizeof(float));
            hipMalloc(&m_v_previous_gpu, g_point_count * g_point_count * sizeof(float));
        }

        Fluid::~Fluid() {
            hipFree(m_density_gpu);
            hipFree(m_density_previous_gpu);
            hipFree(m_u_gpu);
            hipFree(m_v_gpu);
            hipFree(m_u_previous_gpu);
            hipFree(m_v_previous_gpu);
        }

        void Fluid::add_density(float x, float y, float multiplier) {
            if (x <= 0 || x >= 1 || y <= 0 || y >= 1)
                return;
            const auto grid_position = screen_to_array_indices(x, y);
            std::cout << grid_position.i << " " << grid_position.j << "\n";
            add_around_point_kernel<<<g_blocks, g_threads>>>(m_density_gpu, grid_position.i, grid_position.j, multiplier * g_particle_input, 0.03);
            hipDeviceSynchronize();
        }

        void Fluid::add_velocity(float x, float y, float dx, float dy) {
            if (x <= 0 || x >= 1 || y <= 0 || y >= 1) {
                return;
            }
            const auto grid_position = screen_to_array_indices(x, y);
            add_around_point_kernel<<<g_blocks, g_threads>>>(m_u_previous_gpu, grid_position.i, grid_position.j, g_force_input * dx, 0.03);
            add_around_point_kernel<<<g_blocks, g_threads>>>(m_v_previous_gpu, grid_position.i, grid_position.j, g_force_input * dy, 0.03);
            hipDeviceSynchronize();
        }

        float Fluid::sample_density_at(float x, float y) const {
            return m_density.sample_at(x, y);
        }

        float Fluid::sample_u_at(float x, float y) const {
            return m_u.sample_at(x, y);
        }

        float Fluid::sample_v_at(float x, float y) const {
            return m_v.sample_at(x, y);
        }

        void Fluid::clear_previous() {
            set_to_zero_kernel<<<g_blocks, g_threads>>>(m_density_previous_gpu);
            set_to_zero_kernel<<<g_blocks, g_threads>>>(m_v_previous_gpu);
            set_to_zero_kernel<<<g_blocks, g_threads>>>(m_u_previous_gpu);
            hipDeviceSynchronize();
        }

        void Fluid::clear_current() {
            set_to_zero_kernel<<<g_blocks, g_threads>>>(m_density_gpu);
            set_to_zero_kernel<<<g_blocks, g_threads>>>(m_v_gpu);
            set_to_zero_kernel<<<g_blocks, g_threads>>>(m_u_gpu);
            hipDeviceSynchronize();
        }

        void Fluid::density_step() {
            add_scaled(m_density_gpu, m_density_previous_gpu, g_dt);
            diffuse(m_density_previous_gpu, m_density_gpu, g_dt);
            advect(m_density_gpu, m_density_previous_gpu, m_u_gpu, m_v_gpu);

            decrease_density_kernel<<<g_blocks, g_threads>>>(m_density_gpu);
            hipDeviceSynchronize();
        }

        void Fluid::velocity_step() {
            add_scaled(m_u_gpu, m_u_previous_gpu, g_dt);
            add_scaled(m_v_gpu, m_v_previous_gpu, g_dt);
            diffuse(m_u_previous_gpu, m_u_gpu, g_viscosity_coefficient);
            diffuse(m_v_previous_gpu, m_v_gpu, g_viscosity_coefficient);
            project(m_u_previous_gpu, m_v_previous_gpu, m_u_gpu, m_v_gpu);
            advect(m_u_gpu, m_u_previous_gpu, m_u_previous_gpu, m_v_previous_gpu);
            advect(m_v_gpu, m_v_previous_gpu, m_u_previous_gpu, m_v_previous_gpu);
            project(m_u_gpu, m_v_gpu, m_u_previous_gpu, m_v_previous_gpu);
        }

        void Fluid::step() {
            velocity_step();
            density_step();
            get_from_gpu();
        }

        void Fluid::send_to_gpu() {
            hipMemcpy(m_density_gpu, m_density.data(), g_point_count * g_point_count * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(m_density_previous_gpu, m_density_previous.data(), g_point_count * g_point_count * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(m_u_gpu, m_u.data(), g_point_count * g_point_count * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(m_v_gpu, m_v.data(), g_point_count * g_point_count * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(m_u_previous_gpu, m_u_previous.data(), g_point_count * g_point_count * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(m_v_previous_gpu, m_v_previous.data(), g_point_count * g_point_count * sizeof(float), hipMemcpyHostToDevice);
        }

        void Fluid::get_from_gpu() {
            hipMemcpy(m_density.data(), m_density_gpu, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(m_density_previous.data(), m_density_previous_gpu, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(m_u.data(), m_u_gpu, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(m_v.data(), m_v_gpu, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(m_u_previous.data(), m_u_previous_gpu, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(m_v_previous.data(), m_v_previous_gpu, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
        }

        void Fluid::set_circle() {
            set_circle_kernel<<<g_blocks, g_threads>>>(m_density_gpu, m_u_gpu, m_v_gpu);
        }

        const Matrix& Fluid::u() const {
            return m_u;
        }

        const Matrix& Fluid::v() const {
            return m_v;
        }

        const Matrix& Fluid::density() const {
            return m_density;
        }

        /******************** Static function implementations ************************/

        static float clamp_to_zero_one(float val) {
            return val < 0.0f ? 0.0f : val > 1.0f ? 1.0f : val;
        }

        static Index screen_to_array_indices(float x, float y) {
            const auto i = static_cast<uint16_t>(clamp_to_zero_one(x) * (g_point_count - 1));
            const auto j = static_cast<uint16_t>(clamp_to_zero_one(y) * (g_point_count - 1));
            return {i, j};
        }

        static void advect(float* current, const float* previous, const float* u, const float* v) {
            advect_kernel<<<g_blocks, g_threads>>>(current, previous, u, v, g_dt * static_cast<float>(g_cell_count - 1));
            hipDeviceSynchronize();

            set_bounds_to_zero_kernel<<<g_blocks, g_threads>>>(current);
            hipDeviceSynchronize();
        }

        static void diffuse(float* current, const float* previous, float weight) {
            float ratio = weight * g_diffusion_coefficient * (g_cell_count - 1.0f) * (g_cell_count - 1.0f) / g_point_count;
            for (size_t k = 0; k != 20; ++k) {
                diffuse_kernel<<<g_blocks, g_threads>>>(current, previous, ratio);
                hipDeviceSynchronize();
            }

            set_bounds_to_zero_kernel<<<g_blocks, g_threads>>>(current);
            hipDeviceSynchronize();
        }

        static void add_scaled(float* target, const float* addend, const float multiplier) {
            add_scaled_kernel<<<g_blocks, g_threads>>>(target, addend, multiplier);
            hipDeviceSynchronize();
        }

        static void project(float* u_gpu, float* v_gpu, float* u_previous_gpu, float* v_previous_gpu) {
            set_to_zero_kernel<<<g_blocks, g_threads>>>(u_previous_gpu);
            hipDeviceSynchronize();

            project_step_1_kernel<<<g_blocks, g_threads>>>(v_previous_gpu, u_gpu, v_gpu);
            hipDeviceSynchronize();

            for (size_t k = 0; k != 20; ++k) {
                project_step_2_kernel<<<g_blocks, g_threads>>>(u_previous_gpu);
                hipDeviceSynchronize();

                add_scaled(u_previous_gpu, v_previous_gpu, 0.25);
            }

            project_step_3_kernel<<<g_blocks, g_threads>>>(u_gpu, v_gpu, u_previous_gpu);
            hipDeviceSynchronize();

            set_bounds_to_zero_kernel<<<g_blocks, g_threads>>>(u_gpu);
            set_bounds_to_zero_kernel<<<g_blocks, g_threads>>>(v_gpu);
            hipDeviceSynchronize();
        }

        /******************** CUDA kernel function implementations ************************/

        __device__ int diff(uint16_t a, uint16_t b) {
            return a >= b ? static_cast<int>(a - b) : -static_cast<int>(b - a);
        }

        __device__ float distance_kernel(uint16_t i_1, uint16_t j_1, uint16_t i_2, uint16_t j_2) {
            const int d_i = diff(i_1, i_2);
            const int d_j = diff(j_1, j_2);

            return std::sqrt(static_cast<float>(d_i * d_i + d_j * d_j)) / static_cast<float>(g_point_count);
        }

        __global__ void add_around_point_kernel(float* density, uint16_t i_target, uint16_t j_target, float multiplier, float radius) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || i == g_point_count || j == g_point_count)
                return;

            const auto distance = distance_kernel(i, j, i_target, j_target);
            if (distance > radius)
                return;

            density[index] += (radius - distance) * multiplier / radius;
        }

        __global__ void set_to_zero_kernel(float* data) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            data[index] = 0;
        }

        __global__ void decrease_density_kernel(float* density) {
            unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index < g_n)
                density[index] *= 0.98;
        }

        __device__ uint32_t to_linear_index(uint16_t i, uint16_t j) {
            return i * g_point_count + j;
        }

        __global__ void advect_kernel(float* current, const float* previous, const float* u, const float* v, float ratio) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;

            const float x_temp    = static_cast<float>(i) - ratio * u[index];
            const float y_temp    = static_cast<float>(j) - ratio * v[index];
            const float upper     = static_cast<float>(g_cell_count) - 0.5f;
            const float x_new     = x_temp < 0.5f ? 0.5f : x_temp > upper ? upper : x_temp;
            const float y_new     = y_temp < 0.5f ? 0.5f : y_temp > upper ? upper : y_temp;
            const int   x_new_int = static_cast<int>(x_new);
            const int   y_new_int = static_cast<int>(y_new);
            const float dt_x      = x_new - static_cast<float>(x_new_int);
            const float dt_y      = y_new - static_cast<float>(y_new_int);
            current[index]        = (1 - dt_x) * ((1 - dt_y) * previous[to_linear_index(x_new_int, y_new_int)] + dt_y * previous[to_linear_index(x_new_int, y_new_int + 1)]) +
                             dt_x * ((1 - dt_y) * previous[to_linear_index(x_new_int + 1, y_new_int)] + dt_y * previous[to_linear_index(x_new_int + 1, y_new_int + 1)]);
        }

        __global__ void set_bounds_to_zero_kernel(float* data) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;
            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || i == g_point_count - 1 || j == g_point_count - 1)
                data[index] = 0;
        }

        __device__ float sum_neighbors_kernel(const float* data, uint16_t i, uint16_t j) {
            assert(i > 0 && j > 0);
            return data[to_linear_index(i - 1, j)] + data[to_linear_index(i + 1, j)] + data[to_linear_index(i, j - 1)] + data[to_linear_index(i, j + 1)];
        }

        __global__ void diffuse_kernel(float* current, const float* previous, float ratio) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
                return;

            current[index] = (previous[index] + ratio * sum_neighbors_kernel(current, i, j)) / (1 + 4 * ratio);
        }

        __global__ void add_scaled_kernel(float* target, const float* addend, const float multiplier) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            target[index] += multiplier * addend[index];
        }

        __device__ float horizontal_difference(const float* data, uint16_t i, uint16_t j) {
            return data[to_linear_index(i, j + 1)] - data[to_linear_index(i, j - 1)];
        }

        __device__ float vertical_difference(const float* source, uint16_t i, uint16_t j) {
            return source[to_linear_index(i + 1, j)] - source[to_linear_index(i - 1, j)];
        }

        __global__ void project_step_3_kernel(float* u_current, float* v_current, const float* u_previous) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
                return;

            u_current[to_linear_index(i, j)] -= 0.5f * vertical_difference(u_previous, i, j) / g_cell_length;
            v_current[to_linear_index(i, j)] -= 0.5f * horizontal_difference(u_previous, i, j) / g_cell_length;
        }

        __global__ void set_circle_kernel(float* density, float* u, float* v) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
                return;

            const auto distance = distance_kernel(i, j, g_point_count / 2, g_point_count / 2);
            if (distance < 0.3 && distance > 0.2) {
                density[index] = 420;
                u[index] *= 3;
                v[index] *= 3;
            }
        }

        __global__ void project_step_2_kernel(float* u_previous) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
                return;

            u_previous[to_linear_index(i, j)] = sum_neighbors_kernel(u_previous, i, j) / 4.0f;
        }

        __global__ void project_step_1_kernel(float* v_previous, const float* u_current, const float* v_current) {
            const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= g_n)
                return;

            const unsigned i = index / g_point_count;
            const unsigned j = index % g_point_count;
            if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
                return;

            v_previous[index] = -0.5f * g_cell_length * (vertical_difference(u_current, i, j) + horizontal_difference(v_current, i, j));
        }

    } // namespace fluid
} // namespace app
